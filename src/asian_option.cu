

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void asianOptionKernel(float* d_s, float S0, float K, float T, float r, float sigma, int N, hiprandState* states)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = states[tid];
    hiprand_init(1234, tid, 0, &localState);

    float dt = T/N;
    float drift = exp((r - 0.5 * sigma * sigma) * dt);
    float random;
    float current_price = S0;

    float sum_prices = 0.0;

    for(int i = 0; i < N; i++)
    {
        random = hiprand_normal(&localState);
        current_price *= drift * exp(sigma * sqrt(dt) * random);
        sum_prices += current_price;
    }

    float average_price = sum_prices / N;

    d_s[tid] = max(average_price - K, 0.0f);
}
